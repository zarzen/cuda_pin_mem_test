#include "hip/hip_runtime.h"
#include "sock_cli_serv.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

using namespace trans;

__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

double time_now() {
  auto t = std::chrono::high_resolution_clock::now();
  return t.time_since_epoch().count() / 1e9; // convert to seconds
};

void sock_server() {
  SockServ serv("8888");
  serv._listen();
  char msg[4];
  serv._recv(msg, 4);
  serv._send("SYNC", 4);
}

void sock_cli() {
  SockCli cli("127.0.0.1", "8888");
  // std::string msg = "1234";
  cli._send("1234", 4);
  char r[4];
  cli._recv(r, 4);
}

void computation() {
  int N = 120 * (1 << 20);
  float *x, *y, *d_x, *d_y;
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // sync
  sock_cli();
  hipEventRecord(start);
  double s = time_now();
  // Perform SAXPY on 1M elements
  saxpy<<<(N + 511) / 512, 512>>>(N, 2.0f, d_x, d_y);

  hipEventRecord(stop);

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  double e = time_now();
  printf("chrono dur: %lf, start %lf, end %lf\n", e - s, s, e);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Dur: %f \n", milliseconds);
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i] - 4.0f));
  }

  printf("Max error: %f\n", maxError);
  printf("Effective Bandwidth (GB/s): %f\n", N * 4 * 3 / milliseconds / 1e6);
}



void pin_mem() {
  int N = 120 * (1 << 20);
  float *x, *y, *d_x, *d_y;
  size_t msize = N * sizeof(float);
  x = (float *)malloc(msize);


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  sock_server();
  double s = time_now();
  hipEventRecord(start);
  hipHostRegister(x, msize, 0);

  double e = time_now();
  hipEventRecord(stop);
  printf("(chron dur: %lf, start: %lf, end: %lf\n", e-s, s, e);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Pin mem cost dur: %f\n", milliseconds);
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    printf("usage: ./a <mode>");
  }
  char mode = std::atoi(argv[1]);
  if (mode == 0) {
    // serv 
    printf("server mode\n");
    
    pin_mem();
  } else {
    printf("cli mode\n");
    computation();

  }

}